#include "dist_csr.cuh"
#include "combblas_mmio.h"


DistCSR * make_dist_csr(myLocalCSR * my_local_csr, Partitioning *part,
                        uint64_t global_nrows, uint64_t global_ncols, uint64_t global_nnz)
{

    DistCSR * distCSR = (DistCSR*)(Malloc(sizeof(DistCSR)));
    ProcessGrid *grid = part->proc_grid;

    distCSR->global_nrows = global_nrows;
    distCSR->global_ncols = global_ncols;
    distCSR->global_nnz = global_nnz;

    distCSR->my_local_csr = my_local_csr;
    distCSR->mypart = part;
    distCSR->grid = grid;

    return distCSR;
}


DistCSR * read_dist_csr(const char * fpath, Partitioning *part, PartitionFunction part_f, int transpose, int part_balanced, int testpartitioning, Scrambling * scramble)
{
    // Metadata
    uint64_t * edges, deleted;
    ProcessGrid * grid = part->proc_grid;
    float * values;
    ull nnodes;
    uint64_t global_nnz, global_nrows, global_ncols, ndel;

    // Read in the edgelist
    std::string filename(fpath);
    uint64_t nedges = read_dist_mat(filename,
                                    &edges, &values, 
                                    &global_nrows, &global_ncols,
                                    part, part_f, transpose, scramble);

    if (testpartitioning) {
        if(grid->rank == 0) fprintf(stdout, "WARNING: %s function was called with testpartitioning flag on. This means that the mtx values will be deleted and changed with values required by some correctness tests. If you are not in debug mode, please, call this function without this flag.\n", __func__);
        for (int k=0; k<nedges; k++) {
            values[k] = ENCODE_COO2VAL(edges[2*k], edges[2*k+1]);
        }
    }

    uint64_t total_edges = 0;
    MPI_Allreduce(&nedges, &total_edges, 1, MPI_UNSIGNED_LONG_LONG, MPI_SUM, MPI_COMM_WORLD);

#if DEBUG
    fprintf(stdout, "Process %d has %llu edges\n", grid->rank, nedges);

    if(grid->rank==0) fprintf(stdout, "\n========================== Setting up local CSR ==========================\n");
    fflush(stdout);
    sleep(0.1);
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // Compute local matrix dimensions
    uint64_t loc_nrows, loc_ncols;
    compute_local_matrix_dims(part,
                              global_nrows, global_ncols, 
                              &loc_nrows, &loc_ncols);

    // Build local CSR 
    myLocalCSR * local_csr = (myLocalCSR*)(Malloc(sizeof(myLocalCSR)));
    transpose = !part_balanced && transpose;
    build_local_csr_gpu(part, edges, values, nedges, loc_nrows, loc_ncols, (bool)transpose, local_csr);

#if DEBUG
    if(grid->rank==0) fprintf(stdout, "\n========================== Setting up dist CSR ==========================\n");
    fflush(stdout);
    sleep(0.1);
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // Setup DistCSR
    DistCSR * distCSR = make_dist_csr(local_csr, part, global_nrows, global_ncols, total_edges);

#if DEBUG
    if(grid->rank==0) fprintf(stdout, "\n========================== Done setting up dist CSR ==========================\n");
    fflush(stdout);
    sleep(0.1);
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    return distCSR;
}



bool equals(DistCSR * lhs, DistCSR * rhs)
{
    int _equals = (int)equals(lhs->my_local_csr, rhs->my_local_csr);
    if (_equals==0)
        fprintf(stderr, "Incorrect on rank %d\n", lhs->grid->rank);
    MPI_Allreduce(MPI_IN_PLACE, &_equals, 1, MPI_INT, MPI_LAND, lhs->grid->world_comm);
    return _equals;
}


DistCSR * copy(DistCSR * other, bool set_nzr)
{
    DistCSR * result = (DistCSR*)Malloc(sizeof(DistCSR));

    result->my_local_csr = copy(other->my_local_csr, set_nzr);
    result->global_nrows = other->global_nrows;

    result->global_nrows = other->global_nrows;
    result->global_ncols = other->global_ncols;
    result->global_nnz = other->global_nnz;

    result->mypart = (Partitioning * )malloc(sizeof(Partitioning));
    set_mypart_type(result->mypart, other->mypart->my_part_type, other->mypart->operand_type);
    set_mypart_grid(result->mypart, other->grid);
    result->grid = other->grid; // DistCSR does not own process grid
    return result;
}


DistCSR * move(DistCSR * other)
{
    DistCSR * result = (DistCSR*)(Malloc(sizeof(DistCSR)));
    result->my_local_csr = move(other->my_local_csr);
    result->global_nrows = other->global_nrows;

    result->global_nrows = other->global_nrows;
    result->global_ncols = other->global_ncols;
    result->global_nnz = other->global_nnz;
    result->grid = other->grid; // DistCSR does not own process grid
    result->mypart = other->mypart;
    other->mypart = NULL;
    free(other);

    return result;
}


void prune(DistCSR * A, float tol, hipsparseHandle_t * handle)
{
    A->my_local_csr = prune(A->my_local_csr, tol, handle);
    MPI_Allreduce(&A->my_local_csr->nnz, &A->global_nnz, 1, MPI_UINT64_T, MPI_SUM, A->grid->world_comm);
}


void prune_slow(DistCSR * A, float tol, hipsparseHandle_t * handle)
{
    A->my_local_csr = prune_slow(A->my_local_csr, tol, handle);
    MPI_Allreduce(&A->my_local_csr->nnz, &A->global_nnz, 1, MPI_UINT64_T, MPI_SUM, A->grid->world_comm);
}


void write_stats(DistCSR * distCSR, const char * fname)
{
    myLocalCSR * M = distCSR->my_local_csr;
    ProcessGrid * grid = distCSR->grid;
    int rank = grid->rank;
    int n_stats = 3;

    uint64_t send_buf[n_stats] = {M->nnz, M->nnz_r, M->nrows};
    uint64_t * recv_buf = (rank==0) ? (uint64_t*)(malloc(sizeof(uint64_t) * n_stats * grid->nprocs)) : NULL;
    MPI_Gather(send_buf, n_stats, MPI_UINT64_T, recv_buf, n_stats, MPI_UINT64_T, 0, grid->world_comm);

    if (rank==0)
    {

        /* Write to csv */
        FILE * f = fopen(fname, "w");
        fprintf(f, "Rank,nnz,nnzr,nrows\n");
        
        for (int i=0; i<grid->nprocs; i++)
        {
            fprintf(f, "%d,");
            for (int j=0; j<n_stats; j++)
            {
                fprintf(f, "%lu,", recv_buf[j + n_stats * i]);
            }
            fprintf(f, "\n");
        }

        fclose(f);
        free(recv_buf);
    }
}



void print_dist_csr_float(DistCSR * distCSR)
{
    int ntask = distCSR->grid->nprocs;

    for (int i=0; i<ntask; i++) {
        if (distCSR->grid->rank == i) {
            fprintf(stdout, "----- Process %d -----\n", distCSR->grid->rank);
            if (distCSR->global_nnz < 260) {
                print_local_csr<float>(distCSR->my_local_csr);
			}
			fprintf(stdout, "Local nrows:%" PRIu64 "Local ncols:%" PRIu64 "Local nnz:%" PRIu64 " \n",
                    distCSR->my_local_csr->nrows, distCSR->my_local_csr->ncols, distCSR->my_local_csr->nnz);
            MPI_Barrier(MPI_COMM_WORLD);
		}
	}

    MPI_Barrier(MPI_COMM_WORLD);
}


void free_dist_csr(DistCSR * distCSR)
{
    if (distCSR != NULL)
    {
        free_local_csr(distCSR->my_local_csr);
        free(distCSR);
    }
}


